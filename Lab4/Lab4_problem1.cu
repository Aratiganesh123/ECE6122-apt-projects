
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>

#define NORTH 0
#define SOUTH 1
#define EAST  2
#define WEST  3

// Kernel function for performing 2D random walks with directions
__global__ void random_walks(int* x_walks, int* y_walks, int num_walks, int num_steps, unsigned int seed) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState state;
    hiprand_init(seed, tid, 0, &state);  // Initialize random number generator state for each thread

    int x = 0;  // Start at the origin on the x-axis
    int y = 0;  // Start at the origin on the y-axis

    for (int step = 0; step < num_steps; step++) {
        int random_direction = hiprand(&state) % 4;  

        if (random_direction == NORTH) {
            y++;
        } else if (random_direction == SOUTH) {
            y--;
        } else if (random_direction == EAST) {
            x++;
        } else if (random_direction == WEST) {
            x--;
        }
    }

    x_walks[tid] = x;
    y_walks[tid] = y;
}

// Calculate the average distance of walkers from the origin
double calculateAverageDistance(int* x_walks, int* y_walks, int num_walks) {
    double total_distance = 0.0;

    for (int i = 0; i < num_walks; i++) {
        double distance = sqrt(x_walks[i] * x_walks[i] + y_walks[i] * y_walks[i]);
        total_distance += distance;
    }

    return total_distance / num_walks;
}

// Allocate memory and perform random walks using standard CUDA memory allocation
double normalMemoryAllocation(int num_walks, int num_steps, unsigned int seed, int threadsPerBlock, int blocksPerGrid) {
   
    int* d_x_walks;
    int* d_y_walks;
    int x_walks[num_walks];
    int y_walks[num_walks];

    // Allocate device memory for x and y coordinates
    hipMalloc((void**)&d_x_walks, num_walks * sizeof(int));
    hipMalloc((void**)&d_y_walks, num_walks * sizeof(int));

    // Launch the random walk kernel
    random_walks<<<blocksPerGrid, threadsPerBlock>>>(d_x_walks, d_y_walks, num_walks, num_steps, seed);

    // Copy the results back to the host
    hipMemcpy(x_walks, d_x_walks, num_walks * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y_walks, d_y_walks, num_walks * sizeof(int), hipMemcpyDeviceToHost);

    double average_distance = calculateAverageDistance(x_walks, y_walks, num_walks);

    // Free device memory
    hipFree(d_x_walks);
    hipFree(d_y_walks);

    return average_distance;
}

// Allocate memory and perform random walks using pinned CUDA memory allocation
double pinnedMemoryAllocation(int num_walks, int num_steps, unsigned int seed, int threadsPerBlock, int blocksPerGrid) {
   
    int* d_x_walks;
    int* d_y_walks;
    int* x_walks;
    int* y_walks;

    // Allocate pinned memory for x and y coordinates on the host
    hipHostMalloc((void**)&x_walks, num_walks * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void**)&y_walks, num_walks * sizeof(int), hipHostMallocDefault);

    // Allocate device memory for x and y coordinates
    hipMalloc((void**)&d_x_walks, num_walks * sizeof(int));
    hipMalloc((void**)&d_y_walks, num_walks * sizeof(int));

    // Launch the random walk kernel
    random_walks<<<blocksPerGrid, threadsPerBlock>>>(d_x_walks, d_y_walks, num_walks, num_steps, seed);

    // Copy the results back to the host
    hipMemcpy(x_walks, d_x_walks, num_walks * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y_walks, d_y_walks, num_walks * sizeof(int), hipMemcpyDeviceToHost);

    double average_distance = calculateAverageDistance(x_walks, y_walks, num_walks);

    // Free device memory
    hipFree(d_x_walks);
    hipFree(d_y_walks);

    // Free pinned memory on the host
    hipHostFree(x_walks);
    hipHostFree(y_walks);

    return average_distance;
}

// Allocate memory and perform random walks using unified CUDA memory allocation
double unifiedMemoryAllocation(int num_walks, int num_steps, unsigned int seed, int threadsPerBlock, int blocksPerGrid) {
    
    int* x_walks;
    int* y_walks;

    // Allocate unified memory for x and y coordinates
    hipMallocManaged((void**)&x_walks, num_walks * num_steps * sizeof(int));
    hipMallocManaged((void**)&y_walks, num_walks * num_steps * sizeof(int));

    // Launch the random walk kernel
    random_walks<<<blocksPerGrid, threadsPerBlock>>>(x_walks, y_walks, num_walks, num_steps, seed);

    // Ensure that the kernel has finished executing
    hipDeviceSynchronize();

    double average_distance = calculateAverageDistance(x_walks, y_walks, num_walks);

    // Free unified memory (no need to differentiate between host and device)
    hipFree(x_walks);
    hipFree(y_walks);

    return average_distance;
}

int main(int argc, char* argv[]) {
    unsigned int seed = time(NULL);
    int numWalkers = 0;
    int totalSteps = 0;

    std::string programName = argv[0];
    std::vector<std::string> arguments(argv + 1, argv + argc);

    for (size_t i = 0; i < arguments.size(); i++) {
        if (arguments[i] == "-W" && i + 1 < arguments.size()) {
            numWalkers = std::stoi(arguments[i + 1]);
        } else if (arguments[i] == "-I" && i + 1 < arguments.size()) {
            totalSteps = std::stoi(arguments[i + 1]);
        } else if (arguments[i] == "-H") {
            std::cerr << "Usage: " << programName << " [-W <numWalkers>] [-I <totalSteps] [-H]" << std::endl;
            return 1;
        }
    }

    // Set default values if no input was given
    if (numWalkers == 0) {
        numWalkers = 1000;
    }
    if (totalSteps == 0) {
        totalSteps = 10000;
    }

    //std::cout << "Number of Walkers: " << numWalkers << ", Total Steps: " << totalSteps << std::endl;

    // Set the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (numWalkers + threadsPerBlock - 1) / threadsPerBlock;

    //Warmup time 
    auto warmup_avg_time = normalMemoryAllocation(numWalkers, totalSteps, seed, threadsPerBlock, blocksPerGrid);

    // Timer for normal memory allocation
    auto start = std::chrono::high_resolution_clock::now();
    auto normal_avg_time = normalMemoryAllocation(numWalkers, totalSteps, seed, threadsPerBlock, blocksPerGrid);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::microseconds elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Output for Normal CUDA memory Allocation
    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << std::setw(4) << "    Time to calculate(microsec): " << elapsed.count() << std::endl;
    std::cout << std::setw(4) << "    Average distance from origin: " << normal_avg_time << std::endl;

    // Timer for pinned memory allocation
    start = std::chrono::high_resolution_clock::now();
    auto pinned_avg_time = pinnedMemoryAllocation(numWalkers, totalSteps, seed, threadsPerBlock, blocksPerGrid);
    end = std::chrono::high_resolution_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Output for Pinned CUDA memory Allocation
    std::cout << "Pinned CUDA memory Allocation:" << std::endl;
    std::cout << std::setw(4) << "    Time to calculate(microsec): " << elapsed.count() << std::endl;
    std::cout << std::setw(4) << "    Average distance from origin: " << pinned_avg_time << std::endl;

    // Timer for unified memory allocation
    start = std::chrono::high_resolution_clock::now();
    auto unified_avg_time = unifiedMemoryAllocation(numWalkers, totalSteps, seed, threadsPerBlock, blocksPerGrid);
    end = std::chrono::high_resolution_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Output for Managed CUDA memory Allocation
    std::cout << "Managed CUDA memory Allocation:" << std::endl;
    std::cout << std::setw(4) << "    Time to calculate(microsec): " << elapsed.count() << std::endl;
    std::cout << std::setw(4) << "    Average distance from origin: " << unified_avg_time << std::endl;

    std::cout << "Bye" << std::endl;

    return 0;
}
